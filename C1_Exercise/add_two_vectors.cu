#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_exercise.cuh"

#include <cstdio>
#include <cstdlib>

__global__ void initWith(float num, float *a, int N)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x + blockDim.x;

	for (int i = index; i < N; i += stride)
	{
		a[i] = num;
	}
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;

	for (int i = index; i < N; i += stride)
	{
		result[i] = a[i] + b[i];
	}
}

void checkElementsAre(float target, float *vector, int N)
{
	for (int i = 0; i < N; i++)
	{
		if (vector[i] != target)
		{
			printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
			exit(1);
		}
	}
	printf("Success! All values calculated correctly.\n");
}

void add_vectors()
{
	int deviceId;
	int numberOfSMs;
	int deviceCount = 0;

	hipError_t addVectorsErr;
	hipError_t asyncErr;
	
	hipGetDeviceCount(&deviceCount);
	printf("Number of Devices: %d\n", deviceCount);

	hipGetDevice(&deviceId);
	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	printf("Device ID: %d\nNumber of SMs: %d\n", deviceId, numberOfSMs);
	hipSetDevice(deviceId);

	const int N = 2 << 24;
	size_t size = N * sizeof(float);
	float *a, *b, *c, *h_c;

	hipMalloc(&a, size);
	hipMalloc(&b, size);
	hipMalloc(&c, size);
	hipHostMalloc(&h_c, size);

	size_t threadsPerBlock = 64;
	size_t numberOfBlocks = 32 * numberOfSMs;
	
	hipStream_t stream1, stream2, stream3;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>> (3, a, N);
	initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>> (4, b, N);
	initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>> (0, c, N);
	
	addVectorsInto<<<numberOfBlocks, threadsPerBlock>>> (c, a, b, N);

	hipMemcpy(h_c, c, size, hipMemcpyDeviceToHost);

	addVectorsErr = hipGetLastError();
	if (addVectorsErr != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(addVectorsErr));
	}

	asyncErr = hipDeviceSynchronize();
	if (asyncErr != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(asyncErr));
	}

	checkElementsAre(7, h_c, N);

	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipStreamDestroy(stream3);

	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipHostFree(h_c);
}